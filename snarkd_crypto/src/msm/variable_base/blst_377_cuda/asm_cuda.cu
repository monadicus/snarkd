#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "types.h"
#include "asm_cuda.h"

__device__ static inline int is_ge_384(const blst_fp left, const blst_fp right) {
    for (int i = 5; i >= 0; --i) {
        if (left[i] < right[i]) {
            return 0;
        } else if (left[i] > right[i]) {
            return 1;
        }
    }
    return 1;
}

__device__ static inline void sub_mod_384_unchecked(blst_fp ret, const blst_fp a, const blst_fp b) {
   asm(
      "sub.cc.u64 %0, %6, %12;\n\t"
      "subc.cc.u64 %1, %7, %13;\n\t"
      "subc.cc.u64 %2, %8, %14;\n\t"
      "subc.cc.u64 %3, %9, %15;\n\t"
      "subc.cc.u64 %4, %10, %16;\n\t"
      "subc.u64 %5, %11, %17;"
      : "=l"(ret[0]),
      "=l"(ret[1]),
      "=l"(ret[2]),
      "=l"(ret[3]),
      "=l"(ret[4]),
      "=l"(ret[5])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5]),
      "l"(b[0]),
      "l"(b[1]),
      "l"(b[2]),
      "l"(b[3]),
      "l"(b[4]),
      "l"(b[5])
    );
    // return cf != 0?
}

__device__ static inline void reduce(blst_fp x, const blst_fp p) {
    if (is_ge_384(x, p)) {
        blst_fp x_sub;
        sub_mod_384_unchecked(x_sub, x, p);
        memcpy(x, x_sub, sizeof(blst_fp));
    }
}

// The Montgomery reduction here is based on Algorithm 14.32 in
// Handbook of Applied Cryptography
// <http://cacr.uwaterloo.ca/hac/about/chap14.pdf>.
__device__ static inline void mont_384(blst_fp ret, limb_t r[12], const blst_fp p, const limb_t p_inv) {
    // printf("c-t%i:0: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    limb_t k = r[0] * p_inv;
    
    limb_t cross_carry = 0;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"

      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 %6, %6, c;\n\t"
      "addc.u64 %7, 0, 0;\n\t"
      "}"
      : "+l"(r[0]),
      "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "=l"(cross_carry)
      : "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:1: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    k = r[1] * p_inv;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"

      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 c, c, %7;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "addc.cc.u64 %6, %6, c;\n\t"
      "addc.u64 %7, nc, 0;\n\t"
      "}"
      : "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(cross_carry)
      : "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:2: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[2] * p_inv;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"
      
      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 c, c, %7;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "addc.cc.u64 %6, %6, c;\n\t"
      "addc.u64 %7, nc, 0;\n\t"
      "}"
      : "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(cross_carry)
      : "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:3: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    k = r[3] * p_inv;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"
      
      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 c, c, %7;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "addc.cc.u64 %6, %6, c;\n\t"
      "addc.u64 %7, nc, 0;\n\t"
      "}"
      : "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(cross_carry)
      : "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:4: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[4] * p_inv;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"
      
      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 c, c, %7;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "addc.cc.u64 %6, %6, c;\n\t"
      "addc.u64 %7, nc, 0;\n\t"
      "}"
      : "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(cross_carry)
      : "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:5: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);
    k = r[5] * p_inv;

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 t;\n\t"
      ".reg .u64 nc;\n\t"
      
      "mad.lo.cc.u64 c, %14, %8, %0;\n\t"
      "madc.hi.cc.u64 c, %14, %8, 0;\n\t"
      
      "addc.cc.u64 t, %1, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %1, %14, %9, t;\n\t"
      "madc.hi.cc.u64 c, %14, %9, nc;\n\t"

      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %14, %10, t;\n\t"
      "madc.hi.cc.u64 c, %14, %10, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %11, t;\n\t"
      "madc.hi.cc.u64 c, %14, %11, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %12, t;\n\t"
      "madc.hi.cc.u64 c, %14, %12, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %13, t;\n\t"
      "madc.hi.cc.u64 c, %14, %13, nc;\n\t"

      "addc.cc.u64 c, c, %7;\n\t"
      // "addc.u64 nc, 0, 0;\n\t" if we dont want to clobber cross_carry we need this
      "add.u64 %6, %6, c;\n\t" // and this to be add.cc
      // "addc.u64 %7, nc, 0;\n\t" and this
      "}"
      : "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(r[11])
      : "l"(cross_carry),
      "l"(p[0]),
      "l"(p[1]),
      "l"(p[2]),
      "l"(p[3]),
      "l"(p[4]),
      "l"(p[5]),
      "l"(k)
    );

    // printf("c-t%i:6: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    memcpy(ret, r + 6, sizeof(limb_t) * 6);
    reduce(ret, p);
}

__device__ void mul_mont_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p, limb_t p_inv) {
    limb_t r[12];

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 nc;\n\t"
      ".reg .u64 t;\n\t"
      
      "mad.lo.cc.u64 %0, %12, %18, 0;\n\t"
      "madc.hi.cc.u64 c, %12, %18, 0;\n\t"
      
      "madc.lo.cc.u64 %1, %12, %19, c;\n\t"
      "madc.hi.cc.u64 c, %12, %19, 0;\n\t"

      "madc.lo.cc.u64 %2, %12, %20, c;\n\t"
      "madc.hi.cc.u64 c, %12, %20, 0;\n\t"

      "madc.lo.cc.u64 %3, %12, %21, c;\n\t"
      "madc.hi.cc.u64 c, %12, %21, 0;\n\t"

      "madc.lo.cc.u64 %4, %12, %22, c;\n\t"
      "madc.hi.cc.u64 c, %12, %22, 0;\n\t"

      "madc.lo.cc.u64 %5, %12, %23, c;\n\t"
      "madc.hi.u64 %6, %12, %23, 0;\n\t"


      "mad.lo.cc.u64 %1, %13, %18, %1;\n\t"
      "madc.hi.cc.u64 c, %13, %18, 0;\n\t"
      
      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %2, %13, %19, t;\n\t"
      "madc.hi.cc.u64 c, %13, %19, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %13, %20, t;\n\t"
      "madc.hi.cc.u64 c, %13, %20, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %13, %21, t;\n\t"
      "madc.hi.cc.u64 c, %13, %21, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %13, %22, t;\n\t"
      "madc.hi.cc.u64 c, %13, %22, nc;\n\t"

      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %13, %23, t;\n\t"
      "madc.hi.u64 %7, %13, %23, nc;\n\t"


      "mad.lo.cc.u64 %2, %14, %18, %2;\n\t"
      "madc.hi.cc.u64 c, %14, %18, 0;\n\t"
      
      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %19, t;\n\t"
      "madc.hi.cc.u64 c, %14, %19, nc;\n\t"
      
      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %20, t;\n\t"
      "madc.hi.cc.u64 c, %14, %20, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %21, t;\n\t"
      "madc.hi.cc.u64 c, %14, %21, nc;\n\t"

      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %14, %22, t;\n\t"
      "madc.hi.cc.u64 c, %14, %22, nc;\n\t"

      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %14, %23, t;\n\t"
      "madc.hi.u64 %8, %14, %23, nc;\n\t"



      "mad.lo.cc.u64 %3, %15, %18, %3;\n\t"
      "madc.hi.cc.u64 c, %15, %18, 0;\n\t"
      
      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %15, %19, t;\n\t"
      "madc.hi.cc.u64 c, %15, %19, nc;\n\t"
      
      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %15, %20, t;\n\t"
      "madc.hi.cc.u64 c, %15, %20, nc;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %15, %21, t;\n\t"
      "madc.hi.cc.u64 c, %15, %21, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %15, %22, t;\n\t"
      "madc.hi.cc.u64 c, %15, %22, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %15, %23, t;\n\t"
      "madc.hi.u64 %9, %15, %23, nc;\n\t"
      



      "mad.lo.cc.u64 %4, %16, %18, %4;\n\t"
      "madc.hi.cc.u64 c, %16, %18, 0;\n\t"
      
      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %16, %19, t;\n\t"
      "madc.hi.cc.u64 c, %16, %19, nc;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %16, %20, t;\n\t"
      "madc.hi.cc.u64 c, %16, %20, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %16, %21, t;\n\t"
      "madc.hi.cc.u64 c, %16, %21, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %16, %22, t;\n\t"
      "madc.hi.cc.u64 c, %16, %22, nc;\n\t"
      
      "addc.cc.u64 t, %9, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %9, %16, %23, t;\n\t"
      "madc.hi.u64 %10, %16, %23, nc;\n\t"
      


      "mad.lo.cc.u64 %5, %17, %18, %5;\n\t"
      "madc.hi.cc.u64 c, %17, %18, 0;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %17, %19, t;\n\t"
      "madc.hi.cc.u64 c, %17, %19, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %17, %20, t;\n\t"
      "madc.hi.cc.u64 c, %17, %20, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %17, %21, t;\n\t"
      "madc.hi.cc.u64 c, %17, %21, nc;\n\t"
      
      "addc.cc.u64 t, %9, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %9, %17, %22, t;\n\t"
      "madc.hi.cc.u64 c, %17, %22, nc;\n\t"
      
      "addc.cc.u64 t, %10, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %10, %17, %23, t;\n\t"
      "madc.hi.u64 %11, %17, %23, nc;\n\t"

      "}"
      : "+l"(r[0]),
      "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(r[11])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5]),
      "l"(b[0]),
      "l"(b[1]),
      "l"(b[2]),
      "l"(b[3]),
      "l"(b[4]),
      "l"(b[5])
    );
    
    mont_384(ret, r, p, p_inv);
}

__device__ void sqr_mont_384(blst_fp ret, const blst_fp a, const blst_fp p, limb_t p_inv) {
    limb_t r[12];

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 nc;\n\t"
      ".reg .u64 t;\n\t"

      "mad.lo.cc.u64 %1, %12, %13, 0;\n\t"
      "madc.hi.cc.u64 c, %12, %13, 0;\n\t"

      "madc.lo.cc.u64 %2, %12, %14, c;\n\t"
      "madc.hi.cc.u64 c, %12, %14, 0;\n\t"

      "madc.lo.cc.u64 %3, %12, %15, c;\n\t"
      "madc.hi.cc.u64 c, %12, %15, 0;\n\t"

      "madc.lo.cc.u64 %4, %12, %16, c;\n\t"
      "madc.hi.cc.u64 c, %12, %16, 0;\n\t"

      "madc.lo.cc.u64 %5, %12, %17, c;\n\t"
      "madc.hi.u64 %6, %12, %17, 0;\n\t"

      "mad.lo.cc.u64 %3, %13, %14, %3;\n\t"
      "madc.hi.cc.u64 c, %13, %14, 0;\n\t"
      
      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %13, %15, t;\n\t"
      "madc.hi.cc.u64 c, %13, %15, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %13, %16, t;\n\t"
      "madc.hi.cc.u64 c, %13, %16, nc;\n\t"

      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %13, %17, t;\n\t"
      "madc.hi.u64 %7, %13, %17, nc;\n\t"



      "mad.lo.cc.u64 %5, %14, %15, %5;\n\t"
      "madc.hi.cc.u64 c, %14, %15, 0;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %14, %16, t;\n\t"
      "madc.hi.cc.u64 c, %14, %16, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %14, %17, t;\n\t"
      "madc.hi.u64 %8, %14, %17, nc;\n\t"




      "mad.lo.cc.u64 %7, %15, %16, %7;\n\t"
      "madc.hi.cc.u64 c, %15, %16, 0;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %15, %17, t;\n\t"
      "madc.hi.u64 %9, %15, %17, nc;\n\t"
      


      "mad.lo.cc.u64 %9, %16, %17, %9;\n\t"
      "madc.hi.u64 %10, %16, %17, 0;\n\t"

      "}"
      : "+l"(r[0]),
      "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(r[11])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5])
    );

    // printf("c-t%i:0: X, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, X\n", threadIdx.x, r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10]);

    r[11] = r[10] >> 63;
    r[10] = (r[10] << 1) | (r[9] >> 63);
    r[9] = (r[9] << 1) | (r[8] >> 63);
    r[8] = (r[8] << 1) | (r[7] >> 63);
    r[7] = (r[7] << 1) | (r[6] >> 63);
    r[6] = (r[6] << 1) | (r[5] >> 63);
    r[5] = (r[5] << 1) | (r[4] >> 63);
    r[4] = (r[4] << 1) | (r[3] >> 63);
    r[3] = (r[3] << 1) | (r[2] >> 63);
    r[2] = (r[2] << 1) | (r[1] >> 63);
    r[1] = r[1] << 1;

    // printf("c-t%i:1: X, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    asm(
      "{\n\t"

      "mad.lo.cc.u64 %0, %12, %12, 0;\n\t"
      "madc.hi.cc.u64 %1, %12, %12, %1;\n\t"

      "madc.lo.cc.u64 %2, %13, %13, %2;\n\t"
      "madc.hi.cc.u64 %3, %13, %13, %3;\n\t"
  
      "madc.lo.cc.u64 %4, %14, %14, %4;\n\t"
      "madc.hi.cc.u64 %5, %14, %14, %5;\n\t"
  
      "madc.lo.cc.u64 %6, %15, %15, %6;\n\t"
      "madc.hi.cc.u64 %7, %15, %15, %7;\n\t"
  
      "madc.lo.cc.u64 %8, %16, %16, %8;\n\t"
      "madc.hi.cc.u64 %9, %16, %16, %9;\n\t"
  
      "madc.lo.cc.u64 %10, %17, %17, %10;\n\t"
      "madc.hi.u64 %11, %17, %17, %11;\n\t"

      "}"
      : "+l"(r[0]),
      "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(r[11])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5])
    );
    // printf("c-t%i:2: %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7], r[8], r[9], r[10], r[11]);

    mont_384(ret, r, p, p_inv);
}


__device__ static inline void add_mod_384_unchecked(blst_fp ret, const blst_fp a, const blst_fp b) {
    asm(
      "add.cc.u64 %0, %6, %12;\n\t"
      "addc.cc.u64 %1, %7, %13;\n\t"
      "addc.cc.u64 %2, %8, %14;\n\t"
      "addc.cc.u64 %3, %9, %15;\n\t"
      "addc.cc.u64 %4, %10, %16;\n\t"
      "addc.u64 %5, %11, %17;"
      : "=l"(ret[0]),
      "=l"(ret[1]),
      "=l"(ret[2]),
      "=l"(ret[3]),
      "=l"(ret[4]),
      "=l"(ret[5])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5]),
      "l"(b[0]),
      "l"(b[1]),
      "l"(b[2]),
      "l"(b[3]),
      "l"(b[4]),
      "l"(b[5])
    );
    // return cf != 0?
}

__device__ void add_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    add_mod_384_unchecked(ret, a, b);

    reduce(ret, p);
    // return cf != 0?
}

__device__ void sub_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    blst_fp added;
    memcpy(added, a, sizeof(blst_fp));
    // printf("pre-sub [%llu, %llu, %llu, %llu, %llu, %llu]\n", added[0], added[1], added[2], added[3], added[4], added[5]);
    if (is_gt_384(b, a)) {
      // printf("sub-preduce [%llu, %llu, %llu, %llu, %llu, %llu] > [%llu, %llu, %llu, %llu, %llu, %llu]\n", b[0], b[1], b[2], b[3], b[4], b[5], added[0], added[1], added[2], added[3], added[4], added[5]);
      add_mod_384_unchecked(added, added, p);
      // printf("sub-postduce [%llu, %llu, %llu, %llu, %llu, %llu]\n", added[0], added[1], added[2], added[3], added[4], added[5]);
    } else {
      // printf("sub-nonduce [%llu, %llu, %llu, %llu, %llu, %llu] <= [%llu, %llu, %llu, %llu, %llu, %llu]\n", b[0], b[1], b[2], b[3], b[4], b[5], added[0], added[1], added[2], added[3], added[4], added[5]);
    }
    sub_mod_384_unchecked(ret, added, b);
    // printf("post-sub [%llu, %llu, %llu, %llu, %llu, %llu]\n", ret[0], ret[1], ret[2], ret[3], ret[4], ret[5]);
    // return cf != 0?
}

__device__ void sub_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b) {
    sub_mod_384_unchecked(ret, a, b);
    // return cf != 0?
}

__device__ void add_mod_384_unsafe(blst_fp ret, const blst_fp a, const blst_fp b) {
    add_mod_384_unchecked(ret, a, b);
    // return cf != 0?
}

__device__ static inline void _rshift_384(blst_fp ret, const blst_fp value) {
    ret[0] = (value[1] << 63) | (value[0] >> 1);
    ret[1] = (value[2] << 63) | (value[1] >> 1);
    ret[2] = (value[3] << 63) | (value[2] >> 1);
    ret[3] = (value[4] << 63) | (value[3] >> 1);
    ret[4] = (value[5] << 63) | (value[4] >> 1);
    ret[5] = value[5] >> 1;
}

__device__ void div_by_2_mod_384(blst_fp ret, const blst_fp a) {
    _rshift_384(ret, a);
}

__device__ void cneg_mod_384(blst_fp ret, const blst_fp a, bool flag, const blst_fp p) {
    // just let the compiler cmov
    if (flag) {
        sub_mod_384(ret, p, a, p);
    } else {
        memcpy(ret, a, 6 * sizeof(limb_t));
    }
}
