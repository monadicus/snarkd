#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include <stdio.h>
#include <stdint.h>

static const uint32_t WINDOW_SIZE = 128;
// static const uint32_t BLST_WIDTH = 253;

extern "C" __global__ void msm6_pixel(blst_p1* bucket_lists, const blst_p1_affine* bases_in, const blst_scalar* scalars, const uint32_t* window_lengths, const uint32_t window_count) {
    limb_t index = threadIdx.x / 64;
    size_t shift = threadIdx.x - (index * 64);
    limb_t mask = (limb_t) 1 << (limb_t) shift;

    blst_p1 bucket;
    memcpy(&bucket, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    uint32_t window_start = WINDOW_SIZE * blockIdx.x;
    uint32_t window_end = window_start + window_lengths[blockIdx.x];

    uint32_t activated_bases[WINDOW_SIZE];
    uint32_t activated_base_index = 0;

    // we delay the actual additions to a second loop because it reduces warp divergence (20% practical gain)
    for (uint32_t i = window_start; i < window_end; ++i) {
        limb_t bit = (scalars[i][index] & mask);
        if (bit == 0) {
            continue;
        }
        activated_bases[activated_base_index++] = i;
    }
    uint32_t i = 0;
    for (; i < (activated_base_index / 2 * 2); i += 2) {
        blst_p1 intermediate;
        blst_p1_add_affines_into_projective(&intermediate, &bases_in[activated_bases[i]], &bases_in[activated_bases[i + 1]]);
        blst_p1_add_projective_to_projective(&bucket, &bucket, &intermediate);
    }
    for (; i < activated_base_index; ++i) {
        blst_p1_add_affine_to_projective(&bucket, &bucket, &(bases_in[activated_bases[i]]));
    }

    memcpy(&bucket_lists[threadIdx.x * window_count + blockIdx.x], &bucket, sizeof(blst_p1));
}

extern "C" __global__ void msm6_collapse_rows(blst_p1* target, const blst_p1* bucket_lists, const uint32_t window_count) {
    blst_p1 temp_target;
    uint32_t base = threadIdx.x * window_count;
    uint32_t term = base + window_count;
    memcpy(&temp_target, &bucket_lists[base], sizeof(blst_p1));

    for (uint32_t i = base + 1; i < term; ++i) {
        blst_p1_add_projective_to_projective(&temp_target, &temp_target, &bucket_lists[i]);
    }
    
    memcpy(&target[threadIdx.x], &temp_target, sizeof(blst_p1));
}
