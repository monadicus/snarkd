#include "hip/hip_runtime.h"
#include "types.h"
#include "asm_cuda.h"
#include "blst_377_ops.h"

extern "C" __global__ void sqr_test(blst_fp ret, const blst_fp a) {
    blst_fp_sqr(ret, a);
}

extern "C" __global__ void add_test(blst_fp ret, const blst_fp* a) {
    blst_fp_add(ret, a[0], a[1]);
}

extern "C" __global__ void add_projective_test(blst_p1* ret, const blst_p1* a) {
    blst_p1_add_projective_to_projective(ret, &a[0], &a[1]);
}

extern "C" __global__ void double_projective_test(blst_p1* ret, const blst_p1* a) {
    blst_p1_double(ret, &a[0]);
}

struct projective_affine {
    blst_p1 projective;
    blst_p1_affine affine;
};

extern "C" __global__ void add_projective_affine_test(blst_p1* ret, const struct projective_affine* a) {
    blst_p1_add_affine_to_projective(ret, &a[0].projective, &a[0].affine);
}

extern "C" __global__ void add_affine_test(blst_p1* ret, const blst_p1_affine* a) {
    blst_p1_add_affines_into_projective(ret, &a[0], &a[1]);
}

extern "C" __global__ void affine_round_trip_test(blst_p1_affine* ret, const blst_p1_affine* a) {
    blst_p1 intermediate;
    blst_p1_add_affine_to_projective(&intermediate, &BLS12_377_ZERO_PROJECTIVE, &a[0]);
    blst_p1_projective_into_affine(ret, &intermediate);
}

extern "C" __global__ void sub_test(blst_fp ret, const blst_fp* a) {
    blst_fp_sub(ret, a[0], a[1]);
}

extern "C" __global__ void mul_test(blst_fp ret, const blst_fp* a) {
    blst_fp_mul(ret, a[0], a[1]);
}

extern "C" __global__ void div2_test(blst_fp ret, const blst_fp a) {
    div_by_2_mod_384(ret, a);
}

extern "C" __global__ void inverse_test(blst_fp ret, const blst_fp a) {
    blst_fp_inverse(ret, a);
}

