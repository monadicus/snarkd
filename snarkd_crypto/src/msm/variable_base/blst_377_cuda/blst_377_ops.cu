#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include "types.h"
#include "asm_cuda.h"
#include <stdio.h>

__device__ const blst_p1 BLS12_377_ZERO_PROJECTIVE = {
  {0},
  {ONE_MONT_P},
  {0}
};

__device__ const blst_p1_affine BLS12_377_ZERO_AFFINE = {
  {0},
  {ONE_MONT_P}
};

__device__ const blst_scalar BLS12_377_R = {
  TO_LIMB_T(0x0a11800000000001), TO_LIMB_T(0x59aa76fed0000001),
  TO_LIMB_T(0x60b44d1e5c37b001), TO_LIMB_T(0x12ab655e9a2ca556)
};

__device__ static inline int is_blst_p1_zero(const blst_p1 *p) {
    return p->Z[0] == 0 &&
        p->Z[1] == 0 &&
        p->Z[2] == 0 &&
        p->Z[3] == 0 &&
        p->Z[4] == 0 &&
        p->Z[5] == 0;
}

__device__ static inline int is_blst_fp_zero(const blst_fp p) {
    return p[0] == 0 &&
        p[1] == 0 &&
        p[2] == 0 &&
        p[3] == 0 &&
        p[4] == 0 &&
        p[5] == 0;
}

__device__ static inline int is_blst_fp_eq(const blst_fp p1, const blst_fp p2) {
    return p1[0] == p2[0] &&
        p1[1] == p2[1] &&
        p1[2] == p2[2] &&
        p1[3] == p2[3] &&
        p1[4] == p2[4] &&
        p1[5] == p2[5];
}

__device__ static inline int is_blst_p1_affine_zero(const blst_p1_affine *p) {
    return p->X[0] == 0 &&
        p->X[1] == 0 &&
        p->X[2] == 0 &&
        p->X[3] == 0 &&
        p->X[4] == 0 &&
        p->X[5] == 0;
}

__device__ static const blst_fp BIGINT_ONE = { 1, 0, 0, 0, 0, 0 };

__device__ void blst_fp_inverse(blst_fp out, const blst_fp in) {
    if (is_blst_fp_zero(in)) {
        // this is really bad
        *((int*)NULL);
    }
    // Guajardo Kumar Paar Pelzl
    // Efficient Software-Implementation of Finite Fields with Applications to
    // Cryptography
    // Algorithm 16 (BEA for Inversion in Fp)

    blst_fp u;
    memcpy(u, in, sizeof(blst_fp));
    blst_fp v;
    memcpy(v, BLS12_377_P, sizeof(blst_fp));
    blst_fp b;
    memcpy(b, BLS12_377_R2, sizeof(blst_fp));
    blst_fp c;
    memset(c, 0, sizeof(blst_fp));

    while (!is_blst_fp_eq(u, BIGINT_ONE) && !is_blst_fp_eq(v, BIGINT_ONE)) {
        // printf("c-t%i-inverse_round: u=%llu v=%llu b=%llu c=%llu\n", threadIdx.x, u[0], v[0], b[0], c[0]);
        while ((u[0] & 1) == 0) {
            // printf("c-t%i-inverse_round_u_start: u=%llu b=%llu\n", threadIdx.x, u[0], b[0]);
            div_by_2_mod_384(u, u);

            if ((b[0] & 1) != 0) {
                blst_fp_add_unsafe(b, b, BLS12_377_P);
            }
            div_by_2_mod_384(b, b);
            // printf("c-t%i-inverse_round_u_stop: u=%llu b=%llu\n", threadIdx.x, u[0], b[0]);
        }

        while ((v[0] & 1) == 0) {
            // printf("c-t%i-inverse_round_v_start: u=%llu b=%llu\n", threadIdx.x, v[0], c[0]);
            div_by_2_mod_384(v, v);

            if ((c[0] & 1) != 0) {
                blst_fp_add_unsafe(c, c, BLS12_377_P);
            }
            div_by_2_mod_384(c, c);
            // printf("c-t%i-inverse_round_v_stop: u=%llu b=%llu\n", threadIdx.x, v[0], c[0]);
        }

        if (is_gt_384(u, v)) {
            blst_fp_sub_unsafe(u, u, v);
            
            blst_fp_sub(b, b, c);
        } else {
            blst_fp_sub_unsafe(v, v, u);

            blst_fp_sub(c, c, b);
        }
    }
    if (is_blst_fp_eq(u, BIGINT_ONE)) {
        memcpy(out, b, sizeof(blst_fp));
    } else {
        memcpy(out, c, sizeof(blst_fp));
    }
}

__device__ void blst_p1_projective_into_affine(blst_p1_affine* out, const blst_p1* in) {
    if (is_blst_p1_zero(in)) {
        memset(out->X, 0, sizeof(blst_fp));
        memcpy(out->Y, BLS12_377_ONE, sizeof(blst_fp));
        //todo: set inf
    } else if (is_blst_fp_eq(in->Z, BLS12_377_ONE)) {
        memcpy(out->X, in->X, sizeof(blst_fp));
        memcpy(out->Y, in->Y, sizeof(blst_fp));
    } else {
        blst_fp z_inv;
        // printf("c-t%i:cinverse-in: %llu\n", threadIdx.x, in->Z[0]);
        blst_fp_inverse(z_inv, in->Z);
        // printf("c-t%i:cinverse-out: %llu\n", threadIdx.x, z_inv[0]);
        blst_fp z_inv_squared;
        blst_fp_sqr(z_inv_squared, z_inv);
        blst_fp_mul(out->X, in->X, z_inv_squared);
        blst_fp_mul(z_inv_squared, z_inv_squared, z_inv);
        blst_fp_mul(out->Y, in->Y, z_inv_squared);
    }
}

__device__ void blst_p1_double(blst_p1* out, const blst_p1* in) {
    if (is_blst_p1_zero(in)) {
        memcpy(out, in, sizeof(blst_p1));
    }

    // Z3 = 2*Y1*Z1
    blst_fp_mul(out->Z, in->Y, in->Z);
    blst_fp_add(out->Z, out->Z, out->Z);

    // A = X1^2
    blst_fp a;
    blst_fp_sqr(a, in->X);
    
    // B = Y1^2
    blst_fp b;
    blst_fp_sqr(b, in->Y);

    // C = B^2
    blst_fp c;
    blst_fp_sqr(c, b);

    // D = 2*((X1+B)^2-A-C)
    blst_fp d;
    blst_fp_add(d, in->X, b);
    blst_fp_sqr(d, d);
    blst_fp_sub(d, d, a);
    blst_fp_sub(d, d, c);
    blst_fp_add(d, d, d);

    // E = 3*A
    blst_fp e;
    blst_fp_add(e, a, a);
    blst_fp_add(e, e, a);

    // F = E^2
    blst_fp f;
    blst_fp_sqr(f, e);

    // X3 = F-2*D
    blst_fp_add(out->X, d, d);
    blst_fp_sub(out->X, f, out->X);

    // Y3 = E*(D-X3)-8*C
    blst_fp_sub(out->Y, d, out->X);
    blst_fp_mul(out->Y, out->Y, e);

    blst_fp c3;
    blst_fp_add(c3, c, c); // 2c
    blst_fp_add(c3, c3, c3); // 4c
    blst_fp_add(c3, c3, c3); // 8c
    blst_fp_sub(out->Y, out->Y, c3);
}

__device__ void blst_p1_double_affine(blst_p1* out, const blst_p1_affine* p) {
    /*
        dbl-2009-l from
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#doubling-dbl-2009-l
    */

    // A = X1^2
    blst_fp A;
    blst_fp_sqr(A, p->X);

    // B = Y1^2
    blst_fp B;
    blst_fp_sqr(B, p->Y);

    // C = B^2
    blst_fp C;
    blst_fp_sqr(C, B);

    // D = 2 * ((X1 + B)^2 - A - C)
    blst_fp X1B;
    blst_fp_add(X1B, p->X, B);
    blst_fp_sqr(X1B, X1B);
    blst_fp_sub(X1B, X1B, A);
    blst_fp_sub(X1B, X1B, C);
    blst_fp D;
    blst_fp_add(D, X1B, X1B);

    // E = 3 * A
    blst_fp E;
    blst_fp_add(E, A, A);
    blst_fp_add(E, E, A);

    // F = E^2
    blst_fp F;
    blst_fp_sqr(F, E);

    // X3 = F - 2*D
    memcpy(out->X, F, sizeof(blst_fp));
    blst_fp_sub(out->X, out->X, D);
    blst_fp_sub(out->X, out->X, D);

    // Y3 = E*(D - X3) - 8*C
    blst_fp C8;
    blst_fp_add(C8, C, C);
    blst_fp_add(C8, C8, C8);
    blst_fp_add(C8, C8, C8);
    blst_fp_sub(D, D, out->X);
    blst_fp_mul(E, E, D);
    blst_fp_sub(out->Y, E, C8);

    // Z3 = 2*Y1
    blst_fp_add(out->Z, p->Y, p->Y);
}

__device__ void blst_p1_add_affine_to_projective(blst_p1 *out, const blst_p1 *p1, const blst_p1_affine *p2) {
    if (is_blst_p1_affine_zero(p2)) {
        memcpy(out, p1, sizeof(blst_p1));
        return;
    }

    if (is_blst_p1_zero(p1)) {
        memcpy(out->X, p2->X, sizeof(blst_fp));
        memcpy(out->Y, p2->Y, sizeof(blst_fp));
        memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        return;
    }
  
    // http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-madd-2007-bl
    // Works for all curves.

    // printf("c-t%llu:add:0 %llu,%llu,%llu -> %llu,%llu\n", threadIdx.x, p1->X[0], p1->Y[0], p1->Z[0], p2->X[0], p2->Y[0]);

    // Z1Z1 = Z1^2
    blst_fp z1z1;
    blst_fp_sqr(z1z1, p1->Z);

    // printf("c-t%llu:add:1 %llu\n", threadIdx.x, z1z1[0]);

    // U2 = X2*Z1Z1
    blst_fp u2;
    blst_fp_mul(u2, p2->X, z1z1);

    // printf("c-t%llu:add:2 %llu\n", threadIdx.x, u2[0]);

    // S2 = Y2*Z1*Z1Z1
    blst_fp s2;
    blst_fp_mul(s2, p2->Y, p1->Z);
    blst_fp_mul(s2, s2, z1z1);

    if (is_blst_fp_eq(p1->X, u2) && is_blst_fp_eq(p1->Y, s2)) {
        blst_p1_double(out, p1);
        return;
    }

    // printf("c-t%llu:add:3 %llu\n", threadIdx.x, s2[0]);

    // printf("c-t%llu:add:pre-4 %llu - %llu\n", threadIdx.x, u2[0], p1->X[0]);
    // H = U2-X1
    blst_fp h;
    blst_fp_sub(h, u2, p1->X);

    // printf("c-t%llu:add:4 %llu\n", threadIdx.x, h[0]);

    // HH = H^2
    blst_fp hh;
    blst_fp_sqr(hh, h);
    // printf("c-t%llu:add:5 %llu\n", threadIdx.x, hh[0]);

    // I = 4*HH
    blst_fp i;
    memcpy(i, hh, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_add(i, i, i);
    // printf("c-t%llu:add:6 %llu\n", threadIdx.x, i[0]);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);
    // printf("c-t%llu:add:7 %llu\n", threadIdx.x, j[0]);

    // r = 2*(S2-Y1)
    blst_fp r;
    blst_fp_sub(r, s2, p1->Y);
    blst_fp_add(r, r, r);
    // printf("c-t%llu:add:8 %llu\n", threadIdx.x, r[0]);

    // V = X1*I
    blst_fp v;
    blst_fp_mul(v, p1->X, i);
    // printf("c-t%llu:add:9 %llu\n", threadIdx.x, v[0]);

    // X3 = r^2 - J - 2*V
    blst_fp_sqr(out->X, r);
    // printf("c-t%llu:add:1X %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5]);
    blst_fp_sub(out->X, out->X, j);
    // printf("c-t%llu:add:2X %llu, %llu, %llu, %llu, %llu, %llu -- %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5], j[0], j[1], j[2], j[3], j[4], j[5]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:3X %llu\n", threadIdx.x, out->X[0]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:4X %llu\n", threadIdx.x, out->X[0]);

    // Y3 = r*(V-X3)-2*Y1*J
    blst_fp_mul(j, p1->Y, j);
    blst_fp_add(j, j, j);
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);
    blst_fp_sub(out->Y, out->Y, j);
    // printf("c-t%llu:add:Y %llu\n", threadIdx.x, out->Y[0]);

    // Z3 = (Z1+H)^2-Z1Z1-HH
    blst_fp_add(out->Z, p1->Z, h);
    blst_fp_sqr(out->Z, out->Z);
    blst_fp_sub(out->Z, out->Z, z1z1);
    blst_fp_sub(out->Z, out->Z, hh);
    // printf("c-t%llu:add:Z %llu\n", threadIdx.x, out->Z[0]);
}


__device__ void blst_p1_add_projective_to_projective(blst_p1 *out, const blst_p1 *p1, const blst_p1 *p2) {
    if (is_blst_p1_zero(p2)) {
        memcpy(out, p1, sizeof(blst_p1));
        return;
    }

    if (is_blst_p1_zero(p1)) {
        memcpy(out, p2, sizeof(blst_p1));
        return;
    }

    int p1_is_affine = is_blst_fp_eq(p1->Z, BLS12_377_ONE);
    int p2_is_affine = is_blst_fp_eq(p2->Z, BLS12_377_ONE);
    //todo: confirm generated ptx here is *okay* for warp divergence
    if (p1_is_affine && p2_is_affine) {
        blst_p1_affine p1_affine;
        memcpy(&p1_affine.X, &p1->X, sizeof(blst_fp));
        memcpy(&p1_affine.Y, &p1->Y, sizeof(blst_fp));
        blst_p1_affine p2_affine;
        memcpy(&p2_affine.X, &p2->X, sizeof(blst_fp));
        memcpy(&p2_affine.Y, &p2->Y, sizeof(blst_fp));
        blst_p1_add_affines_into_projective(out, &p1_affine, &p2_affine);
        return;
    } if (p1_is_affine) {
        blst_p1_affine p1_affine;
        memcpy(&p1_affine.X, &p1->X, sizeof(blst_fp));
        memcpy(&p1_affine.Y, &p1->Y, sizeof(blst_fp));
        blst_p1_add_affine_to_projective(out, p2, &p1_affine);
        return;
    } else if (p2_is_affine) {
        blst_p1_affine p2_affine;
        memcpy(&p2_affine.X, &p2->X, sizeof(blst_fp));
        memcpy(&p2_affine.Y, &p2->Y, sizeof(blst_fp));
        blst_p1_add_affine_to_projective(out, p1, &p2_affine);
        return;
    }
  
    // http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-madd-2007-bl
    // Works for all curves.

    // printf("c-t%llu:add:0 %llu,%llu,%llu -> %llu,%llu\n", threadIdx.x, p1->X[0], p1->Y[0], p1->Z[0], p2->X[0], p2->Y[0]);

    // Z1Z1 = Z1^2
    blst_fp z1z1;
    blst_fp_sqr(z1z1, p1->Z);

    // Z2Z2 = Z2^2
    blst_fp z2z2;
    blst_fp_sqr(z2z2, p2->Z);

    // U1 = X1*Z2Z2
    blst_fp u1;
    blst_fp_mul(u1, p1->X, z2z2);

    // U2 = X2*Z1Z1
    blst_fp u2;
    blst_fp_mul(u2, p2->X, z1z1);

    // S1 = Y1*Z2*Z2Z2
    blst_fp s1;
    blst_fp_mul(s1, p1->Y, p2->Z);
    blst_fp_mul(s1, s1, z2z2);

    // S2 = Y2*Z1*Z1Z1
    blst_fp s2;
    blst_fp_mul(s2, p2->Y, p1->Z);
    blst_fp_mul(s2, s2, z1z1);

    // H = U2-U1
    blst_fp h;
    blst_fp_sub(h, u2, u1);

    // printf("c-t%llu:add:4 %llu\n", threadIdx.x, h[0]);

    // HH = H^2
    blst_fp hh;
    blst_fp_sqr(hh, h);
    // printf("c-t%llu:add:5 %llu\n", threadIdx.x, hh[0]);

    // I = 4*HH
    blst_fp i;
    memcpy(i, hh, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_add(i, i, i);
    // printf("c-t%llu:add:6 %llu\n", threadIdx.x, i[0]);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);
    // printf("c-t%llu:add:7 %llu\n", threadIdx.x, j[0]);

    // r = 2*(S2-S1)
    blst_fp r;
    blst_fp_sub(r, s2, s1);
    blst_fp_add(r, r, r);
    // printf("c-t%llu:add:8 %llu\n", threadIdx.x, r[0]);

    // V = U1*I
    blst_fp v;
    blst_fp_mul(v, u1, i);
    // printf("c-t%llu:add:9 %llu\n", threadIdx.x, v[0]);

    // X3 = r^2 - J - 2*V
    blst_fp_sqr(out->X, r);
    // printf("c-t%llu:add:1X %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5]);
    blst_fp_sub(out->X, out->X, j);
    // printf("c-t%llu:add:2X %llu, %llu, %llu, %llu, %llu, %llu -- %llu, %llu, %llu, %llu, %llu, %llu\n", threadIdx.x, out->X[0], out->X[1], out->X[2], out->X[3], out->X[4], out->X[5], j[0], j[1], j[2], j[3], j[4], j[5]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:3X %llu\n", threadIdx.x, out->X[0]);
    blst_fp_sub(out->X, out->X, v);
    // printf("c-t%llu:add:4X %llu\n", threadIdx.x, out->X[0]);

    // Y3 = r*(V-X3)-2*S1*J
    blst_fp_mul(j, s1, j);
    blst_fp_add(j, j, j);
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);
    blst_fp_sub(out->Y, out->Y, j);
    // printf("c-t%llu:add:Y %llu\n", threadIdx.x, out->Y[0]);

    // Z3 = ((Z1+Z2)^2-Z1Z1-Z2Z2)*H
    blst_fp_add(out->Z, p1->Z, p2->Z);
    blst_fp_sqr(out->Z, out->Z);
    blst_fp_sub(out->Z, out->Z, z1z1);
    blst_fp_sub(out->Z, out->Z, z2z2);
    blst_fp_mul(out->Z, out->Z, h);
    // printf("c-t%llu:add:Z %llu\n", threadIdx.x, out->Z[0]);
}

__device__ void blst_p1_add_affines_into_projective(blst_p1* out, const blst_p1_affine* p1, const blst_p1_affine* p2) {
    /*
        mmadd-2007-bl from
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian-0.html#addition-mmadd-2007-bl
    */

    // The points can't be 0.
    if (is_blst_p1_affine_zero(p2)) {
        memcpy(out->X, p1->X, sizeof(blst_fp));
        memcpy(out->Y, p1->Y, sizeof(blst_fp));

        if (is_blst_p1_affine_zero(p1)) {
            memcpy(out->Z, BLS12_377_ZERO, sizeof(blst_fp));
        } else {
            memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        }

        return;
    } else if (is_blst_p1_affine_zero(p1)) {
        memcpy(out->X, p2->X, sizeof(blst_fp));
        memcpy(out->Y, p2->Y, sizeof(blst_fp));

        if (is_blst_p1_affine_zero(p2)) {
            memcpy(out->Z, BLS12_377_ZERO, sizeof(blst_fp));
        } else {
            memcpy(out->Z, BLS12_377_ONE, sizeof(blst_fp));
        }

        return;
    }

    // mmadd-2007-bl does not support equal values for p1 and p2.
    // If `p1` and `p2` are equal, use the doubling algorithm.
    if(is_blst_fp_eq(p1->X, p2->X) && is_blst_fp_eq(p1->Y, p2->Y)) {
        blst_p1_double_affine(out, p1);
        return;
    }

    // H = X2-X1
    blst_fp h;
    blst_fp_sub(h, p2->X, p1->X);

    // HH = H^2
    // I = 4*HH
    blst_fp i;
    memcpy(i, h, sizeof(blst_fp));
    blst_fp_add(i, i, i);
    blst_fp_sqr(i, i);

    // J = H*I
    blst_fp j;
    blst_fp_mul(j, h, i);

    // r = 2*(Y2-Y1)
    blst_fp r;
    blst_fp_sub(r, p2->Y, p1->Y);
    blst_fp_add(r, r, r);

    // V = X1*I
    blst_fp v;
    blst_fp_mul(v, p1->X, i);

    // X3 = r^2-J-2*V
    blst_fp_sqr(out->X, r);
    blst_fp_sub(out->X, out->X, j);
    blst_fp_sub(out->X, out->X, v);
    blst_fp_sub(out->X, out->X, v);

    // Y3 = r*(V-X3)-2*Y1*J
    blst_fp_sub(out->Y, v, out->X);
    blst_fp_mul(out->Y, out->Y, r);

    blst_fp y1j;
    blst_fp_mul(y1j, p1->Y, j);
    blst_fp_sub(out->Y, out->Y, y1j);
    blst_fp_sub(out->Y, out->Y, y1j);

    // Z3 = 2*H
    blst_fp_add(out->Z, h, h);
}

__device__ void blst_p1_add_affine_to_affine(blst_p1_affine* out, const blst_p1_affine* p1, const blst_p1_affine* p2) {
    /*
        http://www.hyperelliptic.org/EFD/g1p/auto-shortw.html
        x3 = (y2-y1)2/(x2-x1)2-x1-x2
        y3 = (2*x1+x2)*(y2-y1)/(x2-x1)-(y2-y1)3/(x2-x1)3-y1
    */
    blst_fp y_diff;
    blst_fp_sub(y_diff, p2->Y, p1->Y);

    blst_fp y_diff2;
    blst_fp_sqr(y_diff2, y_diff);

    blst_fp x_diff_inv;
    blst_fp_sub(x_diff_inv, p2->X, p1->X);
    blst_fp_inverse(x_diff_inv, x_diff_inv);
    
    blst_fp x_diff_inv2;
    blst_fp_sqr(x_diff_inv2, x_diff_inv);

    blst_fp sum_x;
    blst_fp_add(sum_x, p1->X, p2->X);

    blst_fp_mul(out->X, y_diff2, x_diff_inv2);
    blst_fp_sub(out->X, out->X, sum_x);

    blst_fp_mul(out->Y, y_diff, x_diff_inv);
    blst_fp_mul(out->Y, out->Y, sum_x);
    blst_fp_add(out->Y, out->Y, out->Y);

    blst_fp y_diff3;
    blst_fp_mul(y_diff3, y_diff2, y_diff);

    blst_fp x_diff_inv3;
    blst_fp_mul(x_diff_inv3, x_diff_inv2, x_diff_inv);

    blst_fp j;
    blst_fp_mul(j, y_diff3, x_diff_inv3);
    blst_fp_sub(out->Y, out->Y, j);

    blst_fp_sub(out->Y, out->Y, p1->Y);
}
